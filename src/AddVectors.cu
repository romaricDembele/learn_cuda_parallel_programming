#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "AddVectors.h"
#include <stdio.h>


// Add Two vectors Kernel definition
__global__ void addVectors(int* A, int* B, int* C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

// ADDITION PROGRAM
void addition() {
    // Dimension of the vectors to be added
    int DIM = 8;

    // Declare pointers on the HOST memory for the vectors
    // to be added (A and B) and the vector holding the result
    int* A;
    int* B;
    int* C;

    // Declare pointers on the HOST memory to hold GPU addresses 
    // of the two vectors to be added.
    int* device_A;
    int* device_B;
    int* device_C;

    // Allocate memory on the HOST memory heap for the tow 
    // vectors to be added
    A = (int *)malloc(DIM * sizeof(int));
    B = (int *)malloc(DIM * sizeof(int));
    C = (int *)malloc(DIM * sizeof(int));

    // Allocate memory on the DEVICE memory heap for the tow 
    // vectors to be added
    hipMalloc((void **)&device_A, DIM * sizeof(int));
    hipMalloc((void **)&device_B, DIM * sizeof(int));
    hipMalloc((void **)&device_C, DIM * sizeof(int));

    // Initialize the vectors to be added in the HOST memory
    for (int i = 0; i < DIM; i++) {
        A[i] = i;
        B[i] = DIM - i;
    }

    // Copy the vectors to the DEVICE memory
    hipMemcpy(device_A, A, DIM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_B, B, DIM * sizeof(int), hipMemcpyHostToDevice);

    // Call the Addition Kernel function
    addVectors<<<1, DIM>>>(device_A, device_B, device_C);

    // Copy the result back to the HOST memory
    hipMemcpy(C, device_C, DIM * sizeof(DIM), hipMemcpyDeviceToHost);
 
    // Ensure that the CPU wait for the GPU workers to finish
    // their jobs.
    hipDeviceSynchronize();

    // Print on the standard output
    for (int i = 0; i < DIM; i++) {
        printf("%d + %d = %d\n", A[i], B[i], C[i]);
    }

    // Free the memory both for the GPU and the CPU
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_B);
    free(A);
    free(B);
    free(C);
}