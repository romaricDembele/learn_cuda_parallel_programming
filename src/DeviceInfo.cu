#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "DeviceInfo.h"

void getDeviceInfo() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess) {
        printf("Error getting device count: %s\n", hipGetErrorString(error));
        return;
    }

    printf("\nNumber of GPUs: %d.\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        printf("GPU n°%d name = %s.\n", i, prop.name);
        printf("GPU n°%d threads/block max number = %d.\n", i, prop.maxThreadsPerBlock);
        
    }
    
}